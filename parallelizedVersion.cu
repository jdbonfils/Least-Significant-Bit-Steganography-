#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
extern "C" {
#include "ppm_lib.h"
}
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <math.h>
#include "string.h"
//Defini la taille du filtre (Sa dimension peut-etre de 3,5,7,9,11.....)
#define DIMFILTRE 5
#define rebord 6
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
#define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif
//Insere un pixel (tab de deux cases) dans un tableau � l'indice indiqu� et decale toute les valeurs  se trouvant � droite de l'indice vers la droite
void insererDansTableauTrie(long *tab, int tailleTab, long tab2[2], int indice)
{
	int temp0;
	int temp1;
	for (int y = indice; y != tailleTab; y = y + 2)
	{
		temp0 = tab[y];
		tab[y] = tab2[0];
		tab2[0] = temp0;

		temp1 = tab[y + 1];
		tab[y + 1] = tab2[1];
		tab2[1] = temp1;
	}
}
//insere dans un tableau tri� le pixel au bonne endroit et r�alise les d�calages n�c�ssaires
//Le tab est un tab 2D en 1D la premiere valeurs est la position du pixels et la deuxieme son poids
void rangerPixelDansTab(long *tab, long tab2[2], int tailleTab)
{
	for (int i = tailleTab - 2; i >= -2; i = i - 2)
	{
		if (tab[i + 1] >= tab2[1])
		{
			if (i == tailleTab - 2)
			{
				break;
			}
			insererDansTableauTrie(tab, tailleTab, tab2, i + 2);
			break;
		}
		if (i == 0)
		{
			insererDansTableauTrie(tab, tailleTab, tab2, i);
		}
	}
}

//La taille du filtre est modifiable, il faut aussi chang� DIMFILTRE plus haut
__constant__ int filtre[DIMFILTRE*DIMFILTRE] = { 5,5,4,2,5,4,7,8,2,1,5,4,0,1,2,4,5,7,5,4,8,4,5,4,6 };

//Kernel permettant de calculer les Vij
//On prend autant de block que de ligne dans l'image, est 1024 thread. Chaque thread traitent 1 ou plusieurs pixels
__global__ void calculVijsSharedMemory(PPMPixel* tabPixels, long* valeurs, int* tailleImageX)
{
	//On est oblig� de recuperer la taille de l'image car blockDim.x ne corespond pas forc�ment a la taille de l'image en X
	//Puisque dans le cas o� l'image � une taille en x superieur � 1024 on ne peux pas prendre autant de thread que de pixel en largeur
	//threadIdx.x correspond au numero de la colonne et  blockIdx.x au numero de la ligne
	int TID = threadIdx.x + blockIdx.x * (*tailleImageX);
	int index = threadIdx.x;
	//Dans la version avec la memoire partag�, pour chaque block on met en m�moire partag� seulement les pixels qui seront utiles au calculs de Vij pour la ligne courante
	//On ne connait pas � l'avance la taille de ce tableau
	extern __shared__ PPMPixel pixelsProche[];
	//Chaque thread du block renseigne une colonne du tableau
	int indexmp = index;
	int TIDtmp = TID;
	while (indexmp < (*tailleImageX)) {
		for (int i = 0; i != DIMFILTRE; i++)
		{
			pixelsProche[indexmp + (i* (*tailleImageX))] = tabPixels[TIDtmp + (i* (*tailleImageX)) - ((DIMFILTRE / 2)*(*tailleImageX))];
		}
		indexmp += 1024;
		TIDtmp += 1024;
	}
	//On attends que tout les threads aient fini leurs travails
	__syncthreads();
	
	while (index < (*tailleImageX)) {
		//Si le numero du thread ne correspond  pas � un pixel sur les bords 
		if (index >= rebord && index < (*tailleImageX) - rebord && (blockIdx.x >= rebord) && blockIdx.x < (gridDim.x) - (rebord)) {
			//Calcul de V(i,j)
			int indiceFiltre = 0;
			for (int b = (-DIMFILTRE / 2); b != (DIMFILTRE / 2) + 1; b++) {
				for (int y = (-DIMFILTRE / 2); y != (DIMFILTRE / 2) + 1; y++) {
					int numeroPixel = index + ((DIMFILTRE / 2) * (*tailleImageX)) + (b * (*tailleImageX)) + y;
					valeurs[TID] += filtre[indiceFiltre] * (pixelsProche[numeroPixel].red + pixelsProche[numeroPixel].green);
					indiceFiltre++;
				}
			}
		}
		else
		{
			//V ij egale � 0 sur les bords
			valeurs[TID] = 0;
		}
		//Si l'image a plus de 1024 thread alors il faut continuer � traiter les pixels pas encore trait�s
		//Dans ce cas l� le thread numero id va s'occuper de traiter le pixel id + 1024  (blockDim.x = 1024 )
		index += 1024;
		TID += 1024;
	}
}
//Cache les characteres dans l'image
void cacherChars(PPMImage *img, char c[])
{

	//On recupere la taille de la chaine de chararactere
	int stringLength = strlen(c);
	int tailleTabPixel = 2 * 8 * strlen(c);
	//Tableau recuperant les n pixels les plus lourd
	long *tabPixels = (long *)malloc(tailleTabPixel * sizeof(long));
	//On initialise le tableau � 0
	for (int i = 0; i != tailleTabPixel; i++)
	{
		tabPixels[i] = 0;
	}


	PPMPixel *pixelsList = img->data;
	PPMPixel *dev_pixels;
	long *tabValeur = (long *)malloc(img->x*img->y * sizeof(long));
	long *dev_Valeurs;
	int tailleImgX = img->x;
	int *dev_TailleImgX;

	hipMalloc((void**)&dev_pixels, img->x*img->y * sizeof(PPMPixel));
	hipMalloc((void**)&dev_Valeurs, img->x*img->y * sizeof(long));
	hipMalloc((void**)&dev_TailleImgX, sizeof(int));

	//Copie du tableau de pixels sur le GPU
	hipMemcpy(dev_pixels, pixelsList, img->x*img->y * sizeof(PPMPixel), hipMemcpyHostToDevice);
	hipMemcpy(dev_TailleImgX, &tailleImgX, sizeof(int), hipMemcpyHostToDevice);

	//Lacement du kernel
	calculVijsSharedMemory << <(img->y), 1024 ,img->x*DIMFILTRE*sizeof(PPMPixel)>> > (dev_pixels, dev_Valeurs, dev_TailleImgX);
	
	//Copie du tableau de valeurs du GPU vers le CPU
	hipMemcpy(tabValeur, dev_Valeurs, img->x*img->y * sizeof(long), hipMemcpyDeviceToHost);

	/* liberer la memoire allouee sur le GPU */
	hipFree(dev_pixels);
	hipFree(dev_Valeurs);
	hipFree(dev_TailleImgX);

	//On cherche les n pixels le plus grand Vij
	for (int v = 0; v != (img->x) * (img->y); v++)
	{

		if (tabValeur[v] != 0) {
			long tab[2] = { v,tabValeur[v] };
			rangerPixelDansTab(tabPixels, tab, tailleTabPixel);
		}
	}
	free(tabValeur);
	//Pour chaque caractere � coder : 
	for (int y = 0; y != stringLength; y++)
	{
		int dec = c[y];
		//Pour chaque bit
		for (int i = 0; i < 8; i++)
		{
			if (dec - pow(2, 7 - i) >= 0)
			{
				dec = dec - pow(2, 7 - i);
				//Si le bit a coder est 1 mais le bit de poids faible du bleu du pixel est 0 alors on le change en 1
				if (img->data[tabPixels[(i * 2) + (y * 8 * 2)]].blue % 2 == 0)
				{
					img->data[tabPixels[(i * 2) + (y * 8 * 2)]].blue += 1;

				}
			}
			else
			{
				//Si le bit a coder est 0 mais le bit de poids faible du bleu du pixel est 0 alors on le change en 1
				if (img->data[tabPixels[(i * 2) + (y * 8 * 2)]].blue % 2 != 0)
				{
					img->data[tabPixels[(i * 2) + (y * 8 * 2)]].blue -= 1;

				}
			}
		}
	}
}

//Trouve les characteres cach�s dans l'image
void trouverChars(PPMImage *img, int nbChar)
{
	//Initialisation du tableau contenant les n valeurs les plus haute
	int tailleTabPixel = 2 * 8 * nbChar;
	//Ce tableau contient les n valeurs les plus lourdes ainsi que leur position coresspondante dans l'image
	long *tabPixels = (long *)malloc(tailleTabPixel * sizeof(long));
	//On initialise le tableau � 0
	for (int i = 0; i != tailleTabPixel; i++)
	{
		tabPixels[i] = 0;
	}

	PPMPixel *pixelsList = img->data;
	PPMPixel *dev_pixels;
	long *tabValeur = (long *)malloc(img->x*img->y * sizeof(long));
	long *dev_Valeurs;
	int tailleImgX = img->x;
	int *dev_TailleImgX;

	hipMalloc((void**)&dev_pixels, img->x*img->y * sizeof(PPMPixel));
	hipMalloc((void**)&dev_Valeurs, img->x*img->y * sizeof(long));
	hipMalloc((void**)&dev_TailleImgX, sizeof(int));

	//Copie du tableau de pixels sur le GPU
	hipMemcpy(dev_pixels, pixelsList, img->x*img->y * sizeof(PPMPixel), hipMemcpyHostToDevice);
	hipMemcpy(dev_TailleImgX, &tailleImgX, sizeof(int), hipMemcpyHostToDevice);
	//Lacement du kernel
	calculVijsSharedMemory << <(img->y), 1024, img->x*DIMFILTRE * sizeof(PPMPixel) >> > (dev_pixels, dev_Valeurs, dev_TailleImgX);

	//Copie du tableau de valeurs du GPU vers le CPU
	hipMemcpy(tabValeur, dev_Valeurs, img->x*img->y * sizeof(long), hipMemcpyDeviceToHost);
	/* liberer la memoire allouee sur le GPU */
	hipFree(dev_pixels);
	hipFree(dev_Valeurs);
	hipFree(dev_TailleImgX);

	//On cherche les 8 pixels les plus <<lourds>>
	for (int v = 0; v != (img->x) * (img->y); v++)
	{

		if (tabValeur[v] != 0) {
			long tab[2] = { v,tabValeur[v] };
			rangerPixelDansTab(tabPixels, tab, tailleTabPixel);
		}
	}

	free(tabValeur);
	for (int y = 0; y != nbChar; y++)
	{
		//Pour chaque octet
		char* dest = (char *)malloc(8);
		for (int i = 0; i != 8; i++)
		{
			//Si le bit de poids faible de la couleur bleu est 0
			if (img->data[tabPixels[(i * 2) + (y * 8 * 2)]].blue % 2 == 0)
			{
				dest[i] = '0';
			}
			//Si le bit de poids faible de la couleur bleu est 1
			else
			{
				dest[i] = '1';
			}
		}
		char e = strtol(dest, (char **)NULL, 2);
		printf(" Charactere trouve : %c \n", e);
	}
}


int main() {
	hipFree(0);
	PPMImage *image;
	//Ouverture de l'image
	image = readPPM("images/gare.ppm");
	//Chaine de char � cacher
	char c[] = "test cacher un char";
	//On affiche la taille du 
	printf("Largeur : %d hauteur : %d \n", image->x, image->y);
	int nbImage = 1;
	//On cache le char dans l'image
	clock_t d = clock();
	for(int i = 0; i != nbImage ; i++)
		cacherChars(image, c);
	clock_t f = clock();
	double time_taken = double(f - d) / double(CLOCKS_PER_SEC);
	printf("%f \n", time_taken);

	//On recherche les n char dans l'image 
	trouverChars(image, 19);

	return 0;
}


